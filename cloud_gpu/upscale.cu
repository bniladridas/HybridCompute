#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>

__device__ float cubicInterpolate(float p0, float p1, float p2, float p3, float t) {
    return p1 + 0.5f * t * (p2 - p0 + t * (2.0f * p0 - 5.0f * p1 + 4.0f * p2 - p3 + t * (3.0f * (p1 - p2) + p3 - p0)));
}

__global__ void bicubicUpscaleKernel(uchar* input, uchar* output, int in_w, int in_h, int scale) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in_w || y >= in_h) return;

    for (int c = 0; c < 3; c++) {
        for (int i = 0; i < scale; i++) {
            for (int j = 0; j < scale; j++) {
                float gx = (float)x + (float)i / (float)scale;
                float gy = (float)y + (float)j / (float)scale;

                int gxi = (int)gx;
                int gyi = (int)gy;

                float vals[4][4];
                for (int m = -1; m <= 2; m++) {
                    for (int n = -1; n <= 2; n++) {
                        int px = min(max(gxi + m, 0), in_w - 1);
                        int py = min(max(gyi + n, 0), in_h - 1);
                        vals[m + 1][n + 1] = input[(py * in_w + px) * 3 + c];
                    }
                }

                float col[4];
                for (int m = 0; m < 4; m++) {
                    col[m] = cubicInterpolate(vals[m][0], vals[m][1], vals[m][2], vals[m][3], gx - gxi);
                }

                float value = cubicInterpolate(col[0], col[1], col[2], col[3], gy - gyi);
                int out_idx = ((y * scale + j) * (in_w * scale) + (x * scale + i)) * 3 + c;
                output[out_idx] = min(max((int)value, 0), 255);
            }
        }
    }
}

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            return -1; \
        } \
    } while (0)

int main() {
    cv::Mat input = cv::imread("input_tile.jpg");
    if (input.empty()) {
        std::cerr << "Error: Could not load image!" << std::endl;
        return -1;
    }

    int in_w = input.cols;
    int in_h = input.rows;
    int scale = 2;
    int out_w = in_w * scale;
    int out_h = in_h * scale;

    cv::Mat output(out_h, out_w, CV_8UC3);

    uchar *d_input, *d_output;
    size_t input_size = in_w * in_h * 3 * sizeof(uchar);
    size_t output_size = out_w * out_h * 3 * sizeof(uchar);

    CUDA_CHECK(hipMalloc(&d_input, input_size));
    CUDA_CHECK(hipMalloc(&d_output, output_size));

    CUDA_CHECK(hipMemcpy(d_input, input.data, input_size, hipMemcpyHostToDevice));

    dim3 blockDim(16, 16);
    dim3 gridDim((in_w + blockDim.x - 1) / blockDim.x, (in_h + blockDim.y - 1) / blockDim.y);
    bicubicUpscaleKernel<<<gridDim, blockDim>>>(d_input, d_output, in_w, in_h, scale);

    CUDA_CHECK(hipMemcpy(output.data, d_output, output_size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));

    cv::imwrite("output_tile.jpg", output);

    return 0;
}
